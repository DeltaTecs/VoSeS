#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <math.h>
#include "hip/hip_runtime.h"

#define CUDA_CHECK_MALLOC(err, msg)            \
    do {                                       \
        if ((err) != hipSuccess) {            \
            printf("%s: %s\n", msg, hipGetErrorString(err));  \
            return false;                      \
        }                                      \
    } while (0)


#define CUDA_CHECK_COPY(err, d_haystack, msg)  \
    do {                                            \
        if ((err) != hipSuccess) {                 \
            printf("%s: %s\n", msg, hipGetErrorString(err));  \
            hipFree(d_haystack);                   \
            return false;                           \
        }                                           \
    } while (0)


#define BYTES_PER_THREAD 20

#define MAX_NEEDLE_LENGTH 64

__constant__ unsigned char c_needle[MAX_NEEDLE_LENGTH];

// Device function to calculate entropy for a given byte array.
__device__ float calculateEntropy(const unsigned char* data, int len) {
    // Create a local histogram for 256 possible byte values.
    float hist[256] = {0.0f};

    // Count the frequency of each byte in the data array.
    for (int i = 0; i < len; ++i) {
        hist[data[i]] += 1.0f;
    }

    // Calculate the entropy using the formula: -sum(p * log2(p)).
    float entropy = 0.0f;
    for (int i = 0; i < 256; ++i) {
        if (hist[i] > 0.0f) {
            float p = hist[i] / len;
            entropy -= p * log2f(p);
        }
    }
    return entropy;
}


__global__ void search_kernel(const char* d_haystack, const long haystack_length, const long needle_length, const char percentile, unsigned long long* candidates) {

    const unsigned long thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint64_t percentile_index = percentile * blockDim.x * gridDim.x * BYTES_PER_THREAD + thread_index * BYTES_PER_THREAD;

    if (percentile_index + BYTES_PER_THREAD + needle_length > haystack_length) {
        return;
    }

 
    unsigned long long local_candidates[2];
    const float entropyThreshold = 5.37f;
    unsigned char entropyInput[512];
    local_candidates[0] = 0;
    local_candidates[1] = 0;

    for (unsigned int offset = 0; offset < BYTES_PER_THREAD; offset++) {

        uint64_t position = percentile_index + offset;

        // copy entropy iunput
        for (int entropy_index = 0; entropy_index < needle_length; entropy_index++) {
            entropyInput[entropy_index] = d_haystack[position + entropy_index];
        }

        float entropy = calculateEntropy(entropyInput, needle_length);
        local_candidates[entropy > entropyThreshold]++;
    }

    atomicAdd(candidates, local_candidates[1]);
}




bool search_array(const unsigned char* haystack, const uint64_t haystack_length, const unsigned char* needle, const uint64_t needle_length) {
    char *d_haystack = nullptr;
    unsigned long long *d_entropy_candidates = nullptr;

    printf("allocating %lld bytes of device mem for haystack...\n", haystack_length);

    // Allocate device memory for haystack
    hipError_t err = hipMalloc((void**)&d_haystack, haystack_length * sizeof(char));
    CUDA_CHECK_MALLOC(err, "hipMalloc failed for d_haystack");

    // copy needle to constant
    err = hipMemcpyToSymbol(HIP_SYMBOL(c_needle), needle, needle_length * sizeof(char));
    if (err != hipSuccess) {
        printf("hipMemcpyToSymbol failed for c_needle: %s\n", hipGetErrorString(err));
        hipFree(d_haystack);
        return false;
    }
    
    // Allocate device memory for d_entropy_candidates
    err = hipMalloc((void**)&d_entropy_candidates, sizeof(unsigned long long));
    CUDA_CHECK_MALLOC(err, "hipMalloc failed for d_entropy_candidates");

    // Copy the haystack from host to device
    err = hipMemcpy(d_haystack, haystack, haystack_length * sizeof(char), hipMemcpyHostToDevice);
    CUDA_CHECK_COPY(err, d_haystack, "hipMemcpy failed for d_haystack");

    // Set d_entropy_candidates to 0
    err = hipMemset(d_entropy_candidates, 0, sizeof(unsigned long long));
    CUDA_CHECK_MALLOC(err, "hipMemset failed for d_entropy_candidates");

    // Define optimal block and grid dimensions using occupancy calculator
    int min_grid_size = 0, block_size = 0;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, search_kernel, 0, 0);

    // Ensure block size does not exceed device capability
    int max_threads_per_block;
    hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    block_size = min(block_size, max_threads_per_block);
    printf("recommended block size: %d\n", block_size);

    // Define block and grid dimensions
    const int THREADS_PER_BLOCK = 1024;
    int total_threads = (haystack_length + BYTES_PER_THREAD - 1) / (BYTES_PER_THREAD * 100);
    int num_blocks = (total_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    dim3 dim_threads(1024, 0);

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the CUDA kernel
    for (int i = 0; i < 100; i++) {
        search_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_haystack, haystack_length, needle_length, i, d_entropy_candidates);
        printf("\r%d%", i);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nTotal runtime of all kernels: %.3f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Check for launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }

    unsigned long long h_entropy_candidates;
    err = hipMemcpy(&h_entropy_candidates, d_entropy_candidates, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying d_entropy_candidates from device to host: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Entropy candidates: %llu\n", h_entropy_candidates);


    // Free the allocated device memory
    hipFree(d_haystack);
    hipFree(d_entropy_candidates);

    return false;
}