#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include "sha256.h"
#include "../util.h"

#define ROTR(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#define SHR(x, n) ((x) >> (n))

#define Ch(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define Maj(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define Sigma0(x) (ROTR((x), 2) ^ ROTR((x), 13) ^ ROTR((x), 22))
#define Sigma1(x) (ROTR((x), 6) ^ ROTR((x), 11) ^ ROTR((x), 25))
#define sigma0(x) (ROTR((x), 7) ^ ROTR((x), 18) ^ SHR((x), 3))
#define sigma1(x) (ROTR((x), 17) ^ ROTR((x), 19) ^ SHR((x), 10))

__constant__ static const uint32_t c_initial_hash[8] = {
    0x6a09e667,
    0xbb67ae85,
    0x3c6ef372,
    0xa54ff53a,
    0x510e527f,
    0x9b05688c,
    0x1f83d9ab,
    0x5be0cd19
};

__constant__ static const uint32_t c_K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

__device__ void process_block(const unsigned char* d_block, uint32_t* d_state) {
        uint32_t W[64];

        for (char t = 0; t < 16; ++t) {
            W[t] = ((uint32_t)d_block[t * 4] << 24) |
                   ((uint32_t)d_block[t * 4 + 1] << 16) |
                   ((uint32_t)d_block[t * 4 + 2] << 8) |
                   ((uint32_t)d_block[t * 4 + 3]);
        }

        for (char t = 16; t < 64; ++t) {
            W[t] = sigma1(W[t - 2]) + W[t - 7] + sigma0(W[t - 15]) + W[t - 16];
        }

        uint32_t a = d_state[0];
        uint32_t b = d_state[1];
        uint32_t c = d_state[2];
        uint32_t d = d_state[3];
        uint32_t e = d_state[4];
        uint32_t f = d_state[5];
        uint32_t g = d_state[6];
        uint32_t h_i = d_state[7];

        for (char t = 0; t < 64; ++t) {
            uint32_t T1 = h_i + Sigma1(e) + Ch(e, f, g) + c_K[t] + W[t];
            uint32_t T2 = Sigma0(a) + Maj(a, b, c);
            h_i = g;
            g = f;
            f = e;
            e = d + T1;
            d = c;
            c = b;
            b = a;
            a = T1 + T2;
        }

        d_state[0] += a;
        d_state[1] += b;
        d_state[2] += c;
        d_state[3] += d;
        d_state[4] += e;
        d_state[5] += f;
        d_state[6] += g;
        d_state[7] += h_i;
}

__device__ void cuda_sha256(const unsigned char *d_input, short input_len, unsigned char *d_digest) {
    short original_byte_length = input_len;
    short total_required = original_byte_length + 1 + 8;
    short blocks_needed = (total_required + 63) / 64;
    short padded_length = blocks_needed * 64;

    uint64_t length_bits = (uint64_t)original_byte_length * 8;

    uint32_t h[8] = {c_initial_hash[0], c_initial_hash[1],
                     c_initial_hash[2], c_initial_hash[3],
                     c_initial_hash[4], c_initial_hash[5],
                     c_initial_hash[6], c_initial_hash[7]};

    // process main blocks
    int num_blocks = padded_length / 64;
    for (size_t i = 0; i < num_blocks - 1; ++i) {
        const unsigned char *block = d_input + i * 64;
        process_block(block, h);
    }

    // process last block (and possibly an extra block if needed)
    unsigned char last_msg_block[64];
    cuda_array_set_zero(last_msg_block, 64);

    // Compute the number of remaining bytes from the original message
    const int r = original_byte_length % 64;
    // Copy the remaining message bytes (if any) into the last block.
    // (If original_byte_length is a multiple of 64, r will be 0.)
    cuda_array_copy(last_msg_block, d_input + original_byte_length - r, r);
    // Append the mandatory 0x80 byte (which is the 1 bit followed by 7 zeros)
    last_msg_block[r] = 0x80;

    // Now, if there is room in this block for the 8-byte length (i.e. if r <= 55),
    // then we can finish this block; otherwise, we must process this block and create an extra block.
    if (r <= 55) {
        // Append length in bits in the final 8 bytes (big-endian)
        last_msg_block[56] = (length_bits >> 56) & 0xFF;
        last_msg_block[57] = (length_bits >> 48) & 0xFF;
        last_msg_block[58] = (length_bits >> 40) & 0xFF;
        last_msg_block[59] = (length_bits >> 32) & 0xFF;
        last_msg_block[60] = (length_bits >> 24) & 0xFF;
        last_msg_block[61] = (length_bits >> 16) & 0xFF;
        last_msg_block[62] = (length_bits >> 8)  & 0xFF;
        last_msg_block[63] = (length_bits)       & 0xFF;
        // Process this final block
        process_block(last_msg_block, h);
    } else {
        // When r > 55 the current block cannot hold the 64-bit length.
        // First process the block with the padding (without the length).
        process_block(last_msg_block, h);
        // Prepare a new block (all zeros) for the length.
        unsigned char extra_block[64] = {0};
        extra_block[56] = (length_bits >> 56) & 0xFF;
        extra_block[57] = (length_bits >> 48) & 0xFF;
        extra_block[58] = (length_bits >> 40) & 0xFF;
        extra_block[59] = (length_bits >> 32) & 0xFF;
        extra_block[60] = (length_bits >> 24) & 0xFF;
        extra_block[61] = (length_bits >> 16) & 0xFF;
        extra_block[62] = (length_bits >> 8)  & 0xFF;
        extra_block[63] = (length_bits)       & 0xFF;
        // Process the extra block that holds the length.
        process_block(extra_block, h);
    }

    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        uint32_t hi = h[i];
        d_digest[i * 4 + 0] = (hi >> 24) & 0xFF;
        d_digest[i * 4 + 1] = (hi >> 16) & 0xFF;
        d_digest[i * 4 + 2] = (hi >> 8) & 0xFF;
        d_digest[i * 4 + 3] = hi & 0xFF;
    }
}